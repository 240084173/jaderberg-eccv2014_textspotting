/** @file gconv.cu
 ** @brief Convolution block
 ** @author Andrea Vedaldi
 **/

#include "mex.h"
#ifdef ENABLE_GPU
#include "gpu/mxGPUArray.h"
#endif
#include "bits/mexutils.h"
#include "bits/im2col.hpp"

#include <blas.h>
#include <iostream>
#include <assert.h>

#ifdef ENABLE_GPU
#include <hipblas.h>
#endif

/* option codes */
enum {
    opt_stride = 0,
    opt_pad,
    opt_verbose
} ;

/* options */
vlmxOption  options [] = {
    {"Stride",           1,   opt_stride            },
    {"Pad",              1,   opt_pad               },
    {"Verbose",          0,   opt_verbose           },
    {0,                  0,   0                     }
} ;

enum {
    IN_DATA = 0, IN_FILTERS1, IN_FILTERS2, IN_DER, IN_END
} ;

enum {
    OUT_RESULT = 0, OUT_RESULT2, OUT_END
} ;

void convgogo(const float* data_im,
               const int height, const int width,
               const float* filter,
               const int kheight, const int kwidth,
               float* data_out)
{
    int height_out = (height - kheight) + 1;
    int width_out = (width - kwidth) + 1;
    
    /* Max naive convolution */
    for (int y = 0; y < height_out; ++y) {
        for (int x = 0; x < width_out; ++x) {
            int val = 0;
            for (int ky = 0; ky < kheight; ++ky) {
                for (int kx = 0; kx < kwidth; ++kx) {
                    data_out[y*width_out + x] += filter[ky*kwidth + kx] * data_im[(y + ky)*width + (x + kx)];
                }
            }
        }
    }
    
}

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
    mxClassID dataClassID ;
    mxClassID filters1ClassID, filters2ClassID ; ;
    mxClassID derClassID ;
    
    mxArray *resultArray ;
    mxArray *dfiltersArray ;
    mxArray *tempArray ;
    
    size_t height, width, depth, numImages, numOut ;
    size_t filter1Height, filter1Width, numFilters1, filter2Height, filter2Width, numFilters2 ;
    size_t derHeight, derWidth, derDepth, numDerImages ;
    int stride = 1 ;
    int pad = 0 ;
    mwSize dataNumDimensions ;
    mwSize filters1NumDimensions, filters2NumDimensions ;
    mwSize derNumDimensions ;
    mwSize const * dataDimensions ;
    mwSize const * filters1Dimensions ;
    mwSize const * filters2Dimensions ;
    mwSize const * derDimensions ;
    mwSize resultDimensions [4] ;
    mwSize dfiltersDimensions [4] ;
    mwSize tempDimensions [2] ;
    
    
    bool const gpuMode = false ;
    bool backMode = false ;
    
    int verbosity = 0 ;
    int opt ;
    int next = IN_END ;
    mxArray const *optarg ;
    
    /* -------------------------------------------------------------- */
    /*                                            Check the arguments */
    /* -------------------------------------------------------------- */
    
    if (nin < 3) {
        mexErrMsgTxt("The arguments are less than three.") ;
    }
    
    if (nin > 3 && vlmxIsString(in[3],-1)) {
        next = 3 ;
        backMode = 0 ;
    } else {
        backMode = (nin >= 4) ;
    }
    
    while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
        switch (opt) {
            case opt_verbose :
                ++ verbosity ;
                break ;
                
            case opt_stride :
                if (!vlmxIsPlainScalar(optarg) || (stride = (int) *mxGetPr(optarg)) < 1) {
                    mexErrMsgTxt("STRIDE must be a positive integer.") ;
                }
                break ;
                
            case opt_pad :
                if (!vlmxIsPlainScalar(optarg) || (pad = (int) *mxGetPr(optarg)) < 0) {
                    mexErrMsgTxt("PAD must be a non-negative integer.") ;
                }
                break ;
                
            default: break ;
        }
    }
    
    
    if (!mxIsNumeric(in[IN_DATA])) {
        mexErrMsgTxt("DATA must be numeric (note: GPU support not compiled).") ;
    }
    
    if (gpuMode) {
        assert(false) ;
    } else {
        if (!mxIsNumeric(in[IN_FILTERS1])) {
            mexErrMsgTxt("DATA is a CPU array but FILTERS1 is not.") ;
        }
        if (!mxIsNumeric(in[IN_FILTERS2])) {
            mexErrMsgTxt("DATA and FILTERS1 are CPU arrays but FILTERS2 is not.") ;
        }
        dataClassID = mxGetClassID(in[IN_DATA]) ;
        dataNumDimensions = mxGetNumberOfDimensions(in[IN_DATA]) ;
        dataDimensions = mxGetDimensions(in[IN_DATA]) ;
        filters1ClassID = mxGetClassID(in[IN_FILTERS1]) ;
        filters2ClassID = mxGetClassID(in[IN_FILTERS2]) ;
        filters1NumDimensions = mxGetNumberOfDimensions(in[IN_FILTERS1]) ;
        filters2NumDimensions = mxGetNumberOfDimensions(in[IN_FILTERS2]) ;
        filters1Dimensions = mxGetDimensions(in[IN_FILTERS1]) ;
        filters2Dimensions = mxGetDimensions(in[IN_FILTERS2]) ;
        if (backMode) {
            derClassID = mxGetClassID(in[IN_DER]) ;
            derNumDimensions = mxGetNumberOfDimensions(in[IN_DER]) ;
            derDimensions = mxGetDimensions(in[IN_DER]) ;
        }
    }
    
    if (dataClassID != mxSINGLE_CLASS) {
        mexErrMsgTxt("DATA is not of class SINGLE.");
    }
    if (filters1ClassID != mxSINGLE_CLASS) {
        mexErrMsgTxt("FILTERS1 is not of class SINGLE.");
    }
    if (filters2ClassID != mxSINGLE_CLASS) {
        mexErrMsgTxt("FILTERS2 is not of class SINGLE.");
    }
    if (backMode && (derClassID != mxSINGLE_CLASS)) {
        mexErrMsgTxt("DER is not of class SINGLE.");
    }
    
    height = dataDimensions[0] ;
    width = dataDimensions[1] ;
    switch (dataNumDimensions) {
        case 2 : depth = 1 ; numImages = 1 ; break ;
        case 3 : depth = dataDimensions[2] ; numImages = 1 ; break ;
        case 4 : depth = dataDimensions[2] ; numImages = dataDimensions[3] ; break ;
        default:  mexErrMsgTxt("DATA has neither two, three nor four dimensions.") ; break ;
    }
    
    filter1Height = filters1Dimensions[0] ;
    filter1Width = filters1Dimensions[1] ;
    filter2Height = filters2Dimensions[0] ;
    filter2Width = filters2Dimensions[1] ;
    if (filters1NumDimensions != filters2NumDimensions) {
        mexErrMsgTxt("FILTERS1 and FILTERS2 do not have the same number of dimensions.");
    }
    switch (filters1NumDimensions) {
        case 2 : numFilters1 = 1 ; numFilters2 = 1 ; break ;
        case 3 : numFilters1 = filters1Dimensions[2] ; numFilters2 = filters2Dimensions[2] ; break ;
        default:  mexErrMsgTxt("FILTERS1 and FILTERS2 have neither two nor three dimensions.") ; break ;
    }
    
    if (backMode) {
        derHeight = derDimensions[0] ;
        derWidth = derDimensions[1] ;
        switch (derNumDimensions) {
            case 2 : derDepth = 1 ; numDerImages = 1 ; break ;
            case 3 : derDepth = derDimensions[2] ; numDerImages = 1 ; break ;
            case 4 : derDepth = derDimensions[2] ; numDerImages = derDimensions[3] ; break ;
            default:  mexErrMsgTxt("DER has neither two, three, nor four dimensions.") ; break ;
        }
    }
    
    
    /* check the two filterbanks correspond */
    if (numFilters1 != numFilters2) {
        mexErrMsgTxt("FILTERS1 and FILTERS2 do not have the same number of filters.");
    }
    
    /* NOT IMPLEMENTED */
    assert(pad==0);
    assert(stride==1);
    
    numOut = depth*numFilters1 ;
    
    tempDimensions[0] = (height + 2*pad - filter1Height)/stride + 1 ;
    tempDimensions[1] = (width + 2*pad - filter1Width)/stride + 1 ;
    
    if (!backMode) {
        resultDimensions[0] = (tempDimensions[0] + 2*pad - filter2Height)/stride + 1 ;
        resultDimensions[1] = (tempDimensions[1] + 2*pad - filter2Width)/stride + 1 ;
        resultDimensions[2] = numOut ;
        resultDimensions[3] = numImages ;
    } else {
        resultDimensions[0] = height ;
        resultDimensions[1] = width ;
        resultDimensions[2] = depth ;
        resultDimensions[3] = numImages ;
        dfiltersDimensions[0] = filter1Height ;
        dfiltersDimensions[1] = filter1Width ;
        dfiltersDimensions[2] = 1 ;
        dfiltersDimensions[3] = numFilters1 ;
    }
    
    
    /* temp1 is for first filter, temp2 is for 2nd filter */
    
    if (verbosity > 0) {
        double const MB = 1024.0*1024.0 ;
        mexPrintf("gconv: mode %s; %s\n", gpuMode?"gpu":"cpu", backMode?"backward":"forward") ;
        mexPrintf("gconv: stride: %d, pad: %d\n", stride, pad) ;
        mexPrintf("gconv: data: %d x %d x %d x %d [%.1f MB]\n",
                  height, width, depth, numImages,
                  (double)(height*width*depth*numImages*4)/MB) ;
        mexPrintf("gconv: filters1: %d x %d x %d x %d [%.1f MB]\n",
                  filter1Height, filter1Width, 1, numFilters1,
                  (double)(filter1Height*filter1Width*numFilters1*4)/MB) ;
        mexPrintf("gconv: filters2: %d x %d x %d x %d [%.1f MB]\n",
                  filter2Height, filter2Width, 1, numFilters2,
                  (double)(filter2Height*filter2Width*numFilters2*4)/MB) ;
        mexPrintf("gconv: result: %d x %d x %d x %d [%.1f MB]\n",
                  resultDimensions[0], resultDimensions[1], resultDimensions[2], resultDimensions[3],
                  (double)(resultDimensions[0]*resultDimensions[1]*resultDimensions[2]*resultDimensions[3]*4)/MB) ;
        if (backMode) {
            mexPrintf("gconv: der: %d x %d x %d x %d [%.1f MB]\n",
                      derHeight, derWidth, derDepth, numDerImages,
                      (double)(derHeight*derWidth*derDepth*numDerImages*4)/MB) ;
            mexPrintf("gconv: dfilters: %d x %d x %d x %d [%.1f MB]\n",
                      dfiltersDimensions[0], dfiltersDimensions[1], dfiltersDimensions[2], dfiltersDimensions[3],
                      (double)(dfiltersDimensions[0]*dfiltersDimensions[1]*dfiltersDimensions[2]*dfiltersDimensions[3]*4)/MB) ;
        }
        mexPrintf("gconv: temp: %d x %d [%.1f MB]\n",
                  tempDimensions[0], tempDimensions[1],
                  (double)(tempDimensions[0]*tempDimensions[1]*4)/MB) ;
    }
    
    if (backMode) {
        if (derHeight != tempDimensions[0] ||
            derWidth != tempDimensions[1] ||
            derDepth != numFilters1 ||
            numDerImages != numImages)
        {
            mexErrMsgTxt("DER dimensions are incompatible with X and FILTERS.") ;
        }
    }
    
    if (height < filter1Height ||  width < filter1Width) {
        mexErrMsgTxt("FILTERS1 are larger than the DATA.") ;
    }
    if (tempDimensions[0] < filter2Height ||  tempDimensions[1] < filter2Width) {
        mexErrMsgTxt("FILTERS2 are larger than the DATA.") ;
    }
    
    if (filter1Height == 0 || filter1Width == 0 || filter2Height == 0 || filter2Width == 0) {
        mexErrMsgTxt("A dimension of FILTERS is void.") ;
    }
    
    /* -------------------------------------------------------------- */
    /*                                                    Do the work */
    /* -------------------------------------------------------------- */
    
    if (gpuMode) {
        assert(false) ;
    } else {
        tempArray = mxCreateNumericArray(2, tempDimensions,
                                         mxSINGLE_CLASS,
                                         mxREAL) ;
        if (!backMode || nout > 1) {
            resultArray = mxCreateNumericArray(4, resultDimensions,
                                               mxSINGLE_CLASS,
                                               mxREAL) ;
        }
        if (backMode) {
            dfiltersArray = mxCreateNumericArray(4, dfiltersDimensions,
                                                 mxSINGLE_CLASS,
                                                 mxREAL);
        }
    }
    
    for (int image = 0 ; image < numImages ; ++image) {
        for (int channel = 0; channel < depth ; ++channel) {
            
            ptrdiff_t dataImOffset = (width*height*depth) * image ;
            ptrdiff_t dataChanOffset = width * height * channel ;
            ptrdiff_t resImOffset = (resultDimensions[0]*resultDimensions[1]*resultDimensions[2]) * image ;
            char OP_N = 'n' ;
            char OP_T = 't' ;
            
            if (backMode) {
                /* ---------------------------------------------------------- */
                /*                                              Backward mode */
                /* ---------------------------------------------------------- */
                assert(false);
            } else {
                /* ---------------------------------------------------------- */
                /*                                               Forward mode */
                /* ---------------------------------------------------------- */
                
                
                
                for (int filter = 0 ; filter < numFilters2 ; ++filter) {
                    /* convolve the 2nd part of each filter individually */
                    ptrdiff_t filter1Offset = filter1Width*filter1Height*filter;
                    ptrdiff_t filter2Offset = filter2Width*filter2Height*filter;
                    ptrdiff_t resChanOffset = resultDimensions[0]*resultDimensions[1] * (channel*numFilters2 + filter);
                    
                    if (gpuMode) {
                        assert(false);
                    } else {
                        /* Filter 1 */
                        convgogo((float const*)mxGetData(in[IN_DATA]) + dataImOffset + dataChanOffset,
                                  width, height,
                                  (float*)mxGetData(in[IN_FILTERS1]) + filter1Offset,
                                  filter1Width, filter1Height,
                                  (float *)mxGetData(tempArray));
                        /* Filter 2 */
                        convgogo((float const*)mxGetData(tempArray),
                                  tempDimensions[1], tempDimensions[0],
                                  (float*)mxGetData(in[IN_FILTERS2]) + filter2Offset,
                                  filter2Width, filter2Height,
                                  (float*)mxGetData(resultArray) + resImOffset + resChanOffset);
                    }
                    
                }
                
            }
        }
    }
    
    
    /* -------------------------------------------------------------- */
    /*                                                        Cleanup */
    /* -------------------------------------------------------------- */
    if (gpuMode) {
        assert(false) ;
    } else {
        mxDestroyArray(tempArray);
        if (backMode) {
            out[OUT_RESULT] = dfiltersArray ;
            if (nout > 1) { out[OUT_RESULT2] = resultArray ; }
        } else {
            out[OUT_RESULT] = resultArray ;
        }
    }
}
